#include "hip/hip_runtime.h"
#include <stdio.h>
#include <random>
#include <time.h>

#define NUM_PARTICLES 100000
#define NUM_ITERATIONS 100
#define BLOCK_SIZE 16  //Number of threads

#define RANDOM_C 1000
#define RANDOM_V 10

#define VELOCITY_DEC 0.0001

typedef struct {
	float3 pos;
	float3 velocity;
}Particle;

//Update the velocity of a particle given an particle array and an index
__device__ void updateVelocity(Particle* par, int index) {
	par[index].pos.x -= VELOCITY_DEC;
	par[index].pos.y -= VELOCITY_DEC;
	par[index].pos.z -= VELOCITY_DEC;
}

//Update the position of a particle given an particle array and an index
__device__ void updatePos(Particle* par, int index) {
	//par[index].pos = par[index].pos + par[index].velocity;
	par[index].pos = make_float3(par[index].pos.x + par[index].velocity.x,
		par[index].pos.y + par[index].velocity.y, par[index].pos.z + par[index].velocity.z);
}

//Kernal function
__global__ void particleSim(Particle* par, int len, int iterations) {
	const int id = threadIdx.x + blockIdx.x * blockDim.x;
	if (id >= len) return;

	for(int i = 0; i < iterations; i++){
		updateVelocity(par, id);
		updatePos(par, id);
	}	
}

void particleCPU(Particle* par, int len) {
	for (int i = 0; i < len; i++) {
		//update velocity
		par[i].pos.x -= VELOCITY_DEC;
		par[i].pos.y -= VELOCITY_DEC;
		par[i].pos.z -= VELOCITY_DEC;

		//update position
		par[i].pos = make_float3(par[i].pos.x + par[i].velocity.x, 
			par[i].pos.y + par[i].velocity.y, par[i].pos.z + par[i].velocity.z);
	}
}

bool equivalent(Particle* p_cpu, Particle* p_gpu, int len){
	float margin = 0.00001;
	for (int i = 0; i < len; i++) {
		//printf("X: %f %f, Y: %f %f Z: %f %f \n", p_gpu[i].pos.x, p_cpu[i].pos.x, p_gpu[i].pos.y,p_cpu[i].pos.y , p_gpu[i].pos.z, p_cpu[i].pos.z);
		//Check position
		if (fabs(p_gpu[i].pos.x - p_cpu[i].pos.x) > margin ||
			fabs(p_gpu[i].pos.y - p_cpu[i].pos.y) > margin ||
			fabs(p_gpu[i].pos.z - p_cpu[i].pos.z) > margin) {
			return false;
		}
		//Check velocity
		if (fabs(p_gpu[i].velocity.x - p_cpu[i].velocity.x) > margin ||
			fabs(p_gpu[i].velocity.y - p_cpu[i].velocity.y) > margin ||
			fabs(p_gpu[i].velocity.z - p_cpu[i].velocity.z) > margin) {
			return false;
		}
	}
	return true;
}
void runSimulation() {

	//To ensure number of blocks is rounded up 
	dim3 numberOfBlocks((NUM_PARTICLES + BLOCK_SIZE - 1) / BLOCK_SIZE);
	dim3 numberOfThreads(BLOCK_SIZE);

	Particle* particles = (Particle*)malloc(NUM_PARTICLES * sizeof(Particle));

	//Fill random values particles
	srand((unsigned int)time(NULL));
	for (int i = 0; i < NUM_PARTICLES; i++) {
		particles[i].pos.x = ((float)rand() / (float)RAND_MAX) * RANDOM_C;
		particles[i].pos.y = ((float)rand() / (float)RAND_MAX) * RANDOM_C;
		particles[i].pos.z = ((float)rand() / (float)RAND_MAX) * RANDOM_C;
			
		particles[i].velocity.x = ((float)rand() / (float)RAND_MAX) * RANDOM_V;
		particles[i].velocity.y = ((float)rand() / (float)RAND_MAX) * RANDOM_V;
		particles[i].velocity.z = ((float)rand() / (float)RAND_MAX) * RANDOM_V;
	}

	//Store the result from gpu here
	Particle* parallel_results = (Particle*)malloc(NUM_PARTICLES * sizeof(Particle));

	Particle* particles_parallel;
	//Allocate gpu memory
	if (hipMalloc(&particles_parallel, sizeof(Particle) * NUM_PARTICLES) != hipSuccess) {
		printf("Error in cudamalloc 1 \n");
		exit(-1);
	}

	//Transfer to gpu memory
	hipMemcpy(particles_parallel, particles, sizeof(Particle) * NUM_PARTICLES, hipMemcpyHostToDevice);

	
	particleSim << <numberOfBlocks, numberOfThreads >> > (particles_parallel, NUM_PARTICLES, NUM_ITERATIONS);
	hipDeviceSynchronize();
	
	hipMemcpy(parallel_results, particles_parallel, sizeof(Particle) * NUM_PARTICLES, hipMemcpyDeviceToHost);


	//CPU
	clock_t start = clock();
	for (int i = 0; i < NUM_ITERATIONS; i++) {
		//printf("%d\n",i);
		particleCPU(particles, NUM_PARTICLES);
	}	
	double time = (double)(clock() - start) / CLOCKS_PER_SEC;

	printf("CPU done in %f seconds!\n", time);

	bool res = equivalent(particles, parallel_results, NUM_PARTICLES);
	
	hipFree(particles_parallel);
	free(particles);
	free(parallel_results);

	if (res) {
		printf("Comparing the output for each implementation, Correct!\n");
	}
	else {
		printf("Comparing the output for each implementation, Wrong \n");
	}

}
int main() {
	runSimulation();
}
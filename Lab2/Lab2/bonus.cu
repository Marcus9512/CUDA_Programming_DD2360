#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <string.h>
#include <random>
#include <math.h>
#include <time.h>

#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>

#define SEED     921
#define NUM_ITER 1000000000

#define NUM_THREADS 256
#define NUM_BLOCKS 1

__global__ void pi_kernel(hiprandState* states, int *res, int iterations) {
    const int id = threadIdx.x + blockIdx.x * blockDim.x;
    //if (id >= iter) return;

    int seed = id; // different seed per thread
    hiprand_init(seed, id, 0, &states[id]);  // 	Initialize CURAND

	int count = 0;
	for(int i = 0; i < iterations; i++){
        
		double x = hiprand_uniform(&states[id]);
		double y = hiprand_uniform(&states[id]);

		//printf("%f %f\n",x,y);
		double z = sqrt((x * x) + (y * y));

		// Check if point is in unit circle
		if (z <= 1.0)
		{
			count ++;
		}
        
	}
	atomicAdd(res, count);  

}

__global__ void pi_kernel_single_prec(hiprandState* states, int* res, int iterations) {
    const int id = threadIdx.x + blockIdx.x * blockDim.x;
    //if (id >= iter) return;

    int seed = id; // different seed per thread
    hiprand_init(seed, id, 0, &states[id]);  // 	Initialize CURAND

    int count = 0;
    for (int i = 0; i < iterations; i++) {

        float x = hiprand_uniform(&states[id]);
        float y = hiprand_uniform(&states[id]);

        //printf("%f %f\n",x,y);

        float z = sqrt((x * x) + (y * y));

        // Check if point is in unit circle
        if (z <= 1.0)
        {
            count++;
        }
    }
    atomicAdd(res, count);

}

void originalCode() {
    int count = 0;
    double x, y, z, pi;

    srand(SEED); // Important: Multiply SEED by "rank" when you introduce MPI!

    // Calculate PI following a Monte Carlo method
    for (int iter = 0; iter < NUM_ITER; iter++)
    {
        // Generate random (X,Y) points
        x = (double)rand() / (double)RAND_MAX;
        y = (double)rand() / (double)RAND_MAX;
        z = sqrt((x * x) + (y * y));

        // Check if point is in unit circle
        if (z <= 1.0)
        {
            count++;
        }
    }

    // Estimate Pi and display the result
    pi = ((double)count / (double)NUM_ITER) * 4.0;

    printf("The result is %f\n", pi);
}

int gpu_solution(bool singlePrec) {

    int* res = (int*)malloc(sizeof(int));

	dim3 numberOfBlocks(NUM_BLOCKS);
    dim3 numberOfThreads(NUM_THREADS);

	float total_amount_of_threads = NUM_BLOCKS * NUM_THREADS;

    int iterationsPerCudaThread = NUM_ITER / total_amount_of_threads;
    int* cuda_res;

    //init random
    hiprandState* dev_random;
    if (hipMalloc((void**)&dev_random, total_amount_of_threads * sizeof(hiprandState)) != hipSuccess) {
        printf("Error in cudamalloc 1 \n");
        exit(-1);
    }

    if (hipMalloc(&cuda_res, sizeof(int)) != hipSuccess) {
        printf("Error in cudamalloc 2 \n");
        exit(-1);
    }
    	

    hipMemset(cuda_res, 0, sizeof(int));

    if (singlePrec) {
        printf("Using single precission\n");
        pi_kernel_single_prec << <numberOfBlocks, numberOfThreads >> > (dev_random, cuda_res, iterationsPerCudaThread);
    }
    else {
        printf("Using double precission\n");
        pi_kernel << <numberOfBlocks, numberOfThreads >> > (dev_random, cuda_res, iterationsPerCudaThread);
    }
    hipDeviceSynchronize();

    hipMemcpy(res, cuda_res, sizeof(int), hipMemcpyDeviceToHost);

    printf("%d\n",*res);
    // Estimate Pi and display the result

    double pi = ((double)*res / (double)NUM_ITER) * 4.0;

    printf("The result is %f\n", pi);

    hipFree(cuda_res);
	hipFree(dev_random);
    free(res);
    return 0;
}

int main(int argc, char* argv[])
{   
    gpu_solution(true);
    return 0;
}


#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <string.h>
#include <random>
#include <math.h>
#include <time.h>

#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>

#define SEED     921
#define NUM_ITER 1000000000

#define NUM_THREADS 256

__global__ void pi_kernel(hiprandState* states, int *res, int iterations) {
    const int id = threadIdx.x + blockIdx.x * blockDim.x;
    //if (id >= iter) return;

    int seed = id; // different seed per thread
    hiprand_init(seed, id, 0, &states[id]);  // 	Initialize CURAND

	int count = 0;
	for(int i = 0; i < iterations; i++){
		double x = hiprand_uniform(&states[id]);
		double y = hiprand_uniform(&states[id]);

		double z = sqrt((x * x) + (y * y));

		// Check if point is in unit circle
		if (z <= 1.0)
		{
			count ++;
		}
	}

	atomicAdd(res, count);

   

}

void originalCode() {
    int count = 0;
    double x, y, z, pi;

    srand(SEED); // Important: Multiply SEED by "rank" when you introduce MPI!

    // Calculate PI following a Monte Carlo method
    for (int iter = 0; iter < NUM_ITER; iter++)
    {
        // Generate random (X,Y) points
        x = (double)rand() / (double)RAND_MAX;
        y = (double)rand() / (double)RAND_MAX;
        z = sqrt((x * x) + (y * y));

        // Check if point is in unit circle
        if (z <= 1.0)
        {
            count++;
        }
    }

    // Estimate Pi and display the result
    pi = ((double)count / (double)NUM_ITER) * 4.0;

    printf("The result is %f\n", pi);
}

int gpu_solution() {

    int* res = (int*)malloc(sizeof(int));

    int iterationsPerCudaThread = NUM_ITER / NUM_THREADS;
    int* cuda_res;

    dim3 numberOfBlocks(1);
    dim3 numberOfThreads(NUM_THREADS);

    //init random
    hiprandState* dev_random;
    hipMalloc((void**)&dev_random, 1 * NUM_THREADS * sizeof(hiprandState));

    hipMemset(cuda_res, 0, sizeof(int));

    
    pi_kernel << <numberOfBlocks, numberOfThreads >> > (dev_random, cuda_res);    
	hipDeviceSynchronize();

    hipMemcpy(res, cuda_res, sizeof(int), hipMemcpyDeviceToHost);

    // Estimate Pi and display the result
    double pi = ((double)*res / (double)NUM_ITER) * 4.0;

    printf("The result is %f\n", pi);

    hipFree(cuda_res);
    free(res);
    return 0;
}

int main(int argc, char* argv[])
{
   
    gpu_solution();
    return 0;
}

